#include "hip/hip_runtime.h"
//GTX 750, ( 4) Multiprocessors, (192) CUDA Cores/MP: 768 CUDA Cores
// OpenCV and I/O libraries
#include <bits/stdc++.h>
#include <fstream>
#include <opencv2/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace chrono;
using namespace cv;

int total_blocks;
int total_threads;
int all_threats;

Mat OriginalImage;
Mat ResizedImage;

const int output_height = 480;
const int output_width = 720;

unsigned char* h_Original;
unsigned char* h_Resized;

void my_cudaError(hipError_t err, string errorMessage){
    if(err != hipSuccess){
        fprintf(stderr, "\nError: %s", errorMessage, hipGetErrorString(err));
    }
}

__global__ void downSizeImage(const unsigned char *original, unsigned char *resized, int W, int H, int w, int h, int all_threats){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int start = idx * ((h * w + all_threats - 1) / all_threats); 
    int end = min(h * w, (idx + 1) * ((h * w + all_threats - 1) / all_threats));

    for(int i = start; i < end; ++i){
        #pragma unroll
        for(int k = 0; k < 3; ++k){
            *(resized + i*3 + k) = *(original + (((H * (i / w)) / h)*W + ((W * (i % w)) / w))*3 + k);
        }
    }

}

/* Host main routine */
int main(int argc, char** argv){

    // Command line input
    char* input_name = argv[1];
    char* output_name = argv[2];
    char* num_blocks = argv[3];
    char* num_threads = argv[4];
    total_blocks = atoi(num_blocks);
    total_threads = atoi(num_threads);
    all_threats = total_blocks * total_threads;
    string nombre_entrada(input_name);
    nombre_entrada = "../images/" + nombre_entrada;

    ofstream fout;
    fout.open("informe_cuda.txt", ios_base::app);
    hipError_t err = hipSuccess;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Read images
    OriginalImage = imread(nombre_entrada);
    ResizedImage = Mat::zeros(output_height, output_width, CV_8UC3);
    if(!OriginalImage.data){
        return cout << "\nCouldn't open or find the image\n", -1;
    }
    size_t og_size = OriginalImage.cols * OriginalImage.rows * 3 * sizeof(unsigned char);
    size_t re_size = output_height * output_width * 3 * sizeof(unsigned char);

    // Allocate the host variables
    h_Original = (unsigned char*) malloc(og_size);
    h_Resized = (unsigned char*) malloc(re_size);
    if(!h_Original or !h_Resized){
        perror("\nError en el malloc de las imagenes en host");
        exit(-1);
    }

    // Initialize the host input array
    for(int i = 0; i < OriginalImage.rows; ++i){
        for(int j = 0; j < OriginalImage.cols; ++j){
            for(int k = 0; k < 3; ++k){
                *(h_Original + (i*OriginalImage.cols*3 + j*3 + k)) = OriginalImage.at<Vec3b>(i, j)[k];
            }
        }
    }

    // Allocate the device input arrays
    unsigned char* d_Original;
    unsigned char* d_Resized;
    err = hipMalloc((void **)&d_Original, og_size);
    my_cudaError(err, "Fallo el malloc en el device del array de la imagen original");
    err = hipMalloc((void **)&d_Resized, re_size);
    my_cudaError(err, "Fallo el malloc en el device del array de la imagen de salida");    

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    err = hipMemcpy(d_Original, h_Original, og_size, hipMemcpyHostToDevice);
    my_cudaError(err, "Fallo en el memcpy del devie para la imagen original");
    err = hipMemcpy(d_Resized, d_Original, re_size, hipMemcpyHostToDevice);
    my_cudaError(err, "Fallo en el memcpy del devie para la imagen de salida");

    //-------------------------------------- Launch the downsize CUDA Kernel-----------------------------------------
    hipEventRecord(start);
    downSizeImage<<<total_blocks, total_threads>>>(d_Original, d_Resized, OriginalImage.cols, OriginalImage.rows, output_width, output_height, all_threats);
    hipEventRecord(stop);
    //----------------------------------------------------------------------------------------------------------------

    // Copy the device result array in device memory to the host result array in host memory
    err = hipMemcpy(h_Resized, d_Resized, re_size, hipMemcpyDeviceToHost);
    my_cudaError(err, "Fallo al traer la imagen del device");
    for(int i = 0; i < output_height; ++i){
        for(int j = 0; j < output_width; ++j){
            for(int k = 0; k < 3; ++k){
                ResizedImage.at<Vec3b>(i, j)[k] = *(h_Resized + (i*output_width*3 + j*3 + k));
            }
        }
    }
    imwrite(output_name, ResizedImage);

    //Gather cuda time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Free device global memory
    err = hipFree(d_Original);
    my_cudaError(err, "Error al liberar la memoria global de device");
    err = hipFree(d_Resized);
    my_cudaError(err, "Error al liberar la memoria global de device");

    // Free host memory
    free(h_Original);
    free(h_Resized);

    // Prints
    fout << fixed << setprecision(12);
    fout << "----------------------------------------------------------------------------\n";
    fout << "Número de bloques: " << total_blocks << '\n';
    fout << "Número de hilos por bloque: " << total_threads << '\n';
    fout << "Número total de hilos: " << all_threats << '\n';
    fout << "Tiempo de respuesta (CUDA): " << milliseconds / 1000 << '\n';
    fout << "Dimensiones de la imagen de entrada: " << OriginalImage.cols << "," << OriginalImage.rows << "\n";
    fout << "----------------------------------------------------------------------------\n\n";
    return 0;
}