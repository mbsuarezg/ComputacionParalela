#include "hip/hip_runtime.h"
//nvcc test.cu -o x -gencode arch=compute_35,code=compute_35 `pkg-config --cflags --libs opencv`
//./x
#include <bits/stdc++.h>

// OpenCV and I/O libraries
#include <fstream>
#include <opencv2/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int total_threads;

Mat OriginalImage;
Mat ResizedImage;

const int output_height = 480;
const int output_width = 720;

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/* Host main routine */
int main(int argc, char** argv)
{
    char* input_name = argv[1];
    char* output_name = argv[2];
    char* num_threads = argv[3];
    total_threads = atoi(num_threads);
    string nombre_entrada(input_name);
    nombre_entrada = "../images/" + nombre_entrada;

    ofstream fout;
    fout.open("informe.txt", ios_base::app);
    
    OriginalIimage = imread(nombre_entrada);
    ResizedImage = Mat::zeros(output_height, output_width, CV_8UC3);
    if(!OriginalIimage.data){
        return cout << "Couldn't open or find the image\n", -1;
    }
    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host variables
    float *h_A, *h_B, *h_C;
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //-------------------------------------- Launch the Vector Add CUDA Kernel-----------------------------------------
    int threadsPerBlock = 128;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy the device result vector in device memory to the host result vector in host memory
    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Test PASSED\n");

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

