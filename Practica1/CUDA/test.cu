#include "hip/hip_runtime.h"
//nvcc test.cu -o x -gencode arch=compute_35,code=compute_35 `pkg-config --cflags --libs opencv`
//./x
#include <bits/stdc++.h>

// OpenCV and I/O libraries
#include <fstream>
#include <opencv2/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace chrono;
using namespace cv;

int total_blocks;
int total_threads;

Mat OriginalIimage;
Mat ResizedImage;

const int output_height = 480;
const int output_width = 720;

unsigned char* h_Original;
unsigned char* h_Resized;

void my_cudaError(hipError_t err, string errorMessage){
    if(err != hipSuccess){
        fprintf(stderr, "\nError: %s", errorMessage, hipGetErrorString(err));
    }
}

__global__ void downSizeImage(const unsigned char *original, unsigned char *resized, int W, int H, int w, int h, int all_threats){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int val = w / all_threats;
    int start = idx * val;
    int end = (idx + 1) * val;
    
    if(end < w){
        for(int i = start; i < end; ++i){
            for(int j = 0; j < h; ++j){
            
                int xoffset = (idx * H) / h;
                int yoffset = (idx * W) / w;
                #pragma unroll
                for(int k = 0; k < 3; ++k){
                    *(resized + (i*3*output_width + j*3 + k)) = *(original + (xoffset*3*output_width + yoffset*3 + k))                
                }
            }
        }    
    }
}

/* Host main routine */
int main(int argc, char** argv)
{
    //command line input
    char* input_name = argv[1];
    char* output_name = argv[2];
    char* num_blocks = argv[3];
    char* num_threads = argv[4];
    total_blocks = atoi(num_blocks);
    total_threads = atoi(num_threads);
    string nombre_entrada(input_name);
    nombre_entrada = "../images/" + nombre_entrada;

    ofstream fout;
    fout.open("informe.txt", ios_base::app);
    hipError_t err = hipSuccess;
    
    //read images
    OriginalIimage = imread(nombre_entrada);
    ResizedImage = Mat::zeros(output_height, output_width, CV_8UC3);
    if(!OriginalIimage.data){
        return cout << "\nCouldn't open or find the image\n", -1;
    }
    size_t og_size = OriginalIimage.cols * OriginalIimage.rows * 3 * sizeof(unsigned char);
    size_t re_size = output_height * output_width * 3 * sizeof(unsigned char);

    // Allocate the host variables
    h_Original = (unsigned char*) malloc(og_size);
    h_Resized = (unsigned char*) malloc(re_size);
    if(!og_image or !dest_image){
        perror("\nError en el malloc de las imagenes");
        exit(-1);
    }

    // Initialize the host input array
    for(int i = 0; i < OriginalIimage.rows; ++i){
        for(int j = 0; j < OriginalIimage.cols; ++j){
            for(int k = 0; k < 3; ++k){
                *(h_Original + (i*OriginalIimage.cols*3 + j*3 + k)) = OriginalIimage.at<Vec3b>(i, j)[k];
            }
        }
    }

    // Allocate the device input vector A
    unsigned char* d_Original;
    unsigned char* d_Resized;
    err = hipMalloc((void **)&d_Original, OriginalIimage.rows * OriginalIimage.cols * 3 * sizeof (unsigned char));
    my_cudaError(err, "Fallo el malloc en el device de el array de la imagen original");

    err = hipMalloc((void **)&d_Resized, output_height * output_width * 3 * sizeof(unsigned char));
    my_cudaError(err, "Fallo el malloc en el device de el array de la imagen de salida");    

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    err = hipMemcpy(d_Original, h_Original, og_size, hipMemcpyHostToDevice);
    my_cudaError(err, "Fallo en el memcpy del devie para la imagen original");

    err = hipMemcpy(d_Resized, d_Original, re_size, hipMemcpyHostToDevice);
    my_cudaError(err, "Fallo en el memcpy del devie para la imagen de salida");

    //-------------------------------------- Launch the downsize CUDA Kernel-----------------------------------------
    int all_threats = total_blocks * total_threads;
    downSizeImage<<<total_blocks, total_threads>>>(d_Original, OriginalIimage.cols, OriginalIimage.rows, output_width, output_height, all_threats);

    //----------------------------------------------------------------------------------------------------------------

    // Copy the device result array in device memory to the host result array in host memory
    err = hipMemcpy(h_Resized, d_Resized, re_size, hipMemcpyDeviceToHost);
    my_cudaError(err, "Fallo al traer la imagen del device");
    for(int i = 0; i < output_height; ++i){
        for(int j = 0; j < output_width; ++j){
            for(int k = 0; k < 3; ++k){
                ResizedImage.at<Vec3b>(i, j)[k] = *(h_Resized + (i*output_width*3 + j*3 + k));
            }
        }
    }
    imwrite(output_name, ResizedImage);

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

