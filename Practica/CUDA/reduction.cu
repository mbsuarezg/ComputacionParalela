#include "hip/hip_runtime.h"
//GTX 750, ( 4) Multiprocessors, (192) CUDA Cores/MP: 768 CUDA Cores
// OpenCV and I/O libraries
#include <bits/stdc++.h>
#include <fstream>
#include <opencv2/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace chrono;
using namespace cv;

int total_blocks;
int total_threads;

Mat h_Original;
Mat h_Resized;

const int output_height = 480;
const int output_width = 720;

void my_cudaError(hipError_t err, string errorMessage){
    if(err != hipSuccess){
        fprintf(stderr, "\nError: %s", errorMessage, hipGetErrorString(err));
    }
}

__global__ void downSizeImage(const unsigned char *original, unsigned char *resized, int W, int H, int w, int h, int all_threats){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int start = idx * ((h * w + all_threats - 1) / all_threats); 
    int end = min(h * w, (idx + 1) * ((h * w + all_threats - 1) / all_threats));

    for(int i = start; i < end; ++i){
        #pragma unroll
        for(int k = 0; k < 3; ++k){
            *(resized + i*3 + k) = *(original + (((H * (i / w)) / h)*W + ((W * (i % w)) / w))*3 + k);
        }
    }

}

/* Host main routine */
int main(int argc, char** argv){

    // Command line input
    char* input_name = argv[1];
    char* output_name = argv[2];
    char* num_blocks = argv[3];
    char* num_threads = argv[4];
    total_blocks = atoi(num_blocks);
    total_threads = atoi(num_threads);
    string nombre_entrada(input_name);
    nombre_entrada = "../images/" + nombre_entrada;

    ofstream fout;
    fout.open("informe_cuda.txt", ios_base::app);
    hipError_t err = hipSuccess;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Read images
    h_Original = imread(nombre_entrada);
    h_Resized = Mat(output_height, output_width, CV_8UC3);
    if(!h_Original.data){
        return cout << "\nCouldn't open or find the image\n", -1;
    }
    size_t og_size = h_Original.cols * h_Original.rows * 3 * sizeof(unsigned char);
    size_t re_size = output_height * output_width * 3 * sizeof(unsigned char);

    // Allocate the device input arrays
    unsigned char* d_Original;
    unsigned char* d_Resized;
    err = hipMalloc((void **)&d_Original, og_size);
    my_cudaError(err, "Fallo el malloc en el device del array de la imagen original");
    err = hipMalloc((void **)&d_Resized, re_size);
    my_cudaError(err, "Fallo el malloc en el device del array de la imagen de salida");    

    // Copy the host input arrays in host memory to the device input arrays in device memory
    err = hipMemcpy(d_Original, h_Original.ptr(), og_size, hipMemcpyHostToDevice);
    my_cudaError(err, "Fallo en el memcpy del devie para la imagen original");
    err = hipMemcpy(d_Resized, h_Resized.ptr(), re_size, hipMemcpyHostToDevice);
    my_cudaError(err, "Fallo en el memcpy del devie para la imagen de salida");

    //-------------------------------------- Launch the downsize CUDA Kernel-----------------------------------------
    hipEventRecord(start);
    downSizeImage<<<total_blocks, total_threads>>>(d_Original, d_Resized, h_Original.cols, h_Original.rows, output_width, output_height, total_blocks * total_threads);
    hipEventRecord(stop);
    //----------------------------------------------------------------------------------------------------------------

    // Copy the device result array in device memory to the host result array in host memory
    err = hipMemcpy(h_Resized.ptr(), d_Resized, re_size, hipMemcpyDeviceToHost);
    my_cudaError(err, "Fallo al traer la imagen del device");
    imwrite(output_name, h_Resized);

    // Gather cuda time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Free device global memory
    err = hipFree(d_Original);
    my_cudaError(err, "Error al liberar la memoria global de device");
    err = hipFree(d_Resized);
    my_cudaError(err, "Error al liberar la memoria global de device");

    // Prints
    fout << fixed << setprecision(12);
    fout << "----------------------------------------------------------------------------\n";
    fout << "Número de bloques: " << total_blocks << '\n';
    fout << "Número de hilos por bloque: " << total_threads << '\n';
    fout << "Número total de hilos: " << total_blocks * total_threads << '\n';
    fout << "Tiempo de respuesta (CUDA): " << milliseconds / 1000 << '\n';
    fout << "Dimensiones de la imagen de entrada: " << h_Original.cols << "," << h_Original.rows << "\n";
    fout << "----------------------------------------------------------------------------\n\n";
    return 0;
}